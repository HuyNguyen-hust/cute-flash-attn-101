#include "flash_fwd_launch_template.h"

// Explicitly instantiate for bfloat16, hdim64
template<>
void run_mha_fwd_<cutlass::bfloat16_t, 64, true>(Flash_fwd_params &params, hipStream_t stream)
{
    run_mha_fwd_hdim64<cutlass::bfloat16_t, true>(params, stream);
}